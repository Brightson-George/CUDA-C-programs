#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void helloWorld() {
	printf("Hello World\n");
}

int main()
{
	int nx, ny;
	nx = 16;
	ny = 4;
	
	dim3 block(8, 2, 1);
	dim3 grid(nx / block.x, ny / block.y, 1);
	
	helloWorld << <grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}