#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void print_threadIDs() {
	printf("threadIdx_x : %d, threadIdx : %d, threadIdx : %d \n",
		threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {
	int nx, ny;
	nx = 16;
	ny = 16;

	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);

	print_threadIDs << <grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();

	return 0;
}